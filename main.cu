
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
using namespace std;

class params{
public:
  int width, height;
  double r_min, r_max, i_min, i_max;
  int max;
  params(){
    width = 3840, height = 2160;
    r_min = -2.86, r_max = 1.86, i_min = -1.32, i_max = 1.32;
    max = 50;
  }
};

void bmp_write(char * img_data, int width, int height, char * filename){
  int size = height*width*3 + 54;
  ofstream bmp(filename, ios::binary);

  char bmp_header[54] = {'B', 'M',
  (char)size, (char)(size >> 8), (char)(size >> 16), (char)(size >> 24),
  0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, (char)width, (char)(width >> 8),
  0, 0, (char)height,(char)(height >> 8), 0, 0, 1, 0, 24, 0, 0, 0, 0, 0,
  (char)(size - 54), (char)((size - 54) >> 8),(char)((size - 54) >> 16),
  (char)((size - 54) >> 24),0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
  0};

  bmp.write(bmp_header, sizeof bmp_header);
  bmp.write(img_data, size - 54);
  bmp.close();
}

__device__ double iter(double r_c, double i_c, int max){
  double r_t = 0.0, i_t = 0.0;
  double r_t_2, i_t_2;
  for (int i = 0; i < max; i++){
    r_t_2 = r_t * r_t - i_t * i_t;
    i_t_2 = 2 * r_t * i_t;
    r_t = r_t_2 + r_c;
    i_t = i_t_2 + i_c;
    if (r_t*r_t + i_t*i_t > 4) return (double)(i + 1 - log(log(r_t*r_t + i_t*i_t))/log(2.0))/(double)max;
  }
  return 0;
}

__global__ void populate(char * d_img_data, double r_min, double r_max, double i_min, double i_max, int width, int height, int max){
  int x = blockIdx.x*blockDim.x + threadIdx.x;
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  if (x < width && y < height){
    double r = ((double)x/(double)width)*(r_max-r_min)+r_min;
    double i = ((double)y/(double)height)*(i_max-i_min)+i_min;
    double t = iter(r, i, max);
    d_img_data[y*3*width + x*3 + 0] = (char)(255*t);
    d_img_data[y*3*width + x*3 + 1] = (char)(255*t*t);
    d_img_data[y*3*width + x*3 + 2] = (char)(255*t*t*t);
  }
}

void plot_frame(params plot, char * filename){
  /*getting image data*/
  char * h_img_data = new char[plot.width*plot.height*3*sizeof(char)];
  char * d_img_data;
  hipMalloc((void **) &d_img_data, plot.width*plot.height*3*sizeof(char));
  dim3 threads(32, 32, 1);
  dim3 grid(plot.width/threads.x, plot.height/threads.y, 1);
  populate<<<grid, threads>>>(d_img_data, plot.r_min, plot.r_max, plot.i_min, plot.i_max, plot.width, plot.height, plot.max);
  hipMemcpy(h_img_data, d_img_data, plot.width*plot.height*3*sizeof(char), hipMemcpyDeviceToHost);
  hipFree(d_img_data);

  /*write to file*/
  bmp_write(h_img_data, plot.width, plot.height, filename);
  delete[] h_img_data;
}

int main(){
  char * filename = (char *)malloc(20*sizeof(char));
  int num_frames = 1;
  params plot;
  clock_t t1 = clock();
  for (int i = 0; i < num_frames; i++){
    sprintf(filename, "anim/%06d.bmp", i);
    plot_frame(plot, filename);
  }
  clock_t t2 = clock();
  printf("Took %1.6f seconds per frame.\n", (double)(t2-t1)/(CLOCKS_PER_SEC*num_frames));
  delete[] filename;
  return 0;
}
