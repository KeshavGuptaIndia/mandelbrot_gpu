
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
using namespace std;

class params{
private:
  double r_min_ini, r_min_trg;
  double r_max_ini, r_max_trg;
  double i_min_ini, i_min_trg;
  double i_max_ini, i_max_trg;

public:
  int width, height;
  double r_min, r_max, i_min, i_max;
  int max;

  params(){
    r_min_ini = -2.86, r_min_trg = -0.1777;
    r_max_ini = 1.86, r_max_trg = -0.1194;
    i_min_ini = -1.33, i_min_trg = 1.0138;
    i_max_ini = 1.33, i_max_trg = 1.0472;

    width = 1920, height = 1080;
    r_min = r_min_ini, r_max = r_max_ini, i_min = i_min_ini, i_max = i_max_ini;
    max = 50;
  }

  void set_frame_number(int, int);
};

void params::set_frame_number(int n, int max){
  if (n < max){
    double t = 1 - pow(0.001, (double)n/(double)max);
    r_min = r_min_ini +  t*(r_min_trg - r_min_ini);
    r_max = r_max_ini +  t*(r_max_trg - r_max_ini);
    i_min = i_min_ini +  t*(i_min_trg - i_min_ini);
    i_max = i_max_ini +  t*(i_max_trg - i_max_ini);
  }
  else{
    r_min = r_min_trg, r_max = r_max_trg, i_min = i_min_trg, i_max = i_max_trg;
  }
}

void bmp_write(char * img_data, int width, int height, char * filename){
  int size = height*width*3 + 54;
  ofstream bmp(filename, ios::binary);

  char bmp_header[54] = {'B', 'M',
  (char)size, (char)(size >> 8), (char)(size >> 16), (char)(size >> 24),
  0, 0, 0, 0, 54, 0, 0, 0, 40, 0, 0, 0, (char)width, (char)(width >> 8),
  0, 0, (char)height,(char)(height >> 8), 0, 0, 1, 0, 24, 0, 0, 0, 0, 0,
  (char)(size - 54), (char)((size - 54) >> 8),(char)((size - 54) >> 16),
  (char)((size - 54) >> 24),0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
  0};

  bmp.write(bmp_header, sizeof bmp_header);
  bmp.write(img_data, size - 54);
  bmp.close();
}

__host__ __device__ double iter(double r_c, double i_c, int max){
  double r_t = 0.0, i_t = 0.0;
  double r_t_2, i_t_2, mag;
  for (int i = 0; i < max; i++){
    r_t_2 = r_t * r_t - i_t * i_t;
    i_t_2 = 2 * r_t * i_t;
    r_t = r_t_2 + r_c;
    i_t = i_t_2 + i_c;
    mag = r_t*r_t + i_t*i_t;
    if (mag > 4){
      return (double)(i + 1 - log(log(mag))/log(2.0))/(double)max;
    }
  }
  return 0;
}

__global__ void populate(char * d_img_data, double r_min, double r_max,
  double i_min, double i_max, int width, int height, int max){
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    if (x < width && y < height){
      double r = ((double)x/(double)width)*(r_max-r_min)+r_min;
      double i = ((double)y/(double)height)*(i_max-i_min)+i_min;
      double t = iter(r, i, max);
      d_img_data[y*3*width + x*3 + 0] = (char)(255*sqrt(t));
      d_img_data[y*3*width + x*3 + 1] = (char)(255*t);
      d_img_data[y*3*width + x*3 + 2] = (char)(255*t*t);
    }
  }

void plot_frame_gpu(params plot, char * filename){
  /*getting image data*/
  char * h_img_data = new char[plot.width*plot.height*3*sizeof(char)];
  char * d_img_data;
  hipMalloc((void **) &d_img_data, plot.width*plot.height*3*sizeof(char));
  dim3 threads(32, 32, 1);
  dim3 grid(ceil((double)plot.width/(double)threads.x),
    ceil((double)plot.height/(double)threads.y), 1);
  populate<<<grid, threads>>>(d_img_data, plot.r_min, plot.r_max, plot.i_min,
    plot.i_max, plot.width, plot.height, plot.max);
  hipMemcpy(h_img_data, d_img_data, plot.width*plot.height*3*sizeof(char),
    hipMemcpyDeviceToHost);
  hipFree(d_img_data);

  /*write to file*/
  bmp_write(h_img_data, plot.width, plot.height, filename);
  delete[] h_img_data;
}

void plot_frame_cpu(params plot, char * filename){
  char * img_data = new char[plot.width*plot.height*3*sizeof(char)];
  for (int y = 0; y < plot.height; y++){
    for (int x = 0; x < plot.width; x++){
      double r = ((double)x/(double)plot.width)*(plot.r_max-plot.r_min)+plot.r_min;
      double i = ((double)y/(double)plot.height)*(plot.i_max-plot.i_min)+plot.i_min;
      double t = iter(r, i, plot.max);
      img_data[y*3*plot.width + x*3 + 0] = (char)(255*sqrt(t));
      img_data[y*3*plot.width + x*3 + 1] = (char)(255*t);
      img_data[y*3*plot.width + x*3 + 2] = (char)(255*t*t);
    }
  }
  bmp_write(img_data, plot.width, plot.height, filename);
  delete[] img_data;
}

int main(int argc, char ** argv){
  int num_frames = 10;
  clock_t t1, t2;
  char * filename = (char *)malloc(200*sizeof(char));
  params plot;

  /*TEST GPU*/
  ofstream log_gpu("/media/keshav/Keshav/Dropbox (MIT)/anim_gpu/log_gpu.txt");
  for (int i = 0; i < num_frames; i++){
    sprintf(filename, "/media/keshav/Keshav/Dropbox (MIT)/anim_gpu/i0001%02d.bmp", i);
    plot.set_frame_number(i, num_frames);
    t1 = clock();
    plot_frame_gpu(plot, filename);
    t2 = clock();
    log_gpu << 1000*(double)(t2-t1)/(CLOCKS_PER_SEC) << "\n";
    if (!(i%(num_frames/100)) && (argc-1)) printf("Done %2.2f%% on GPU.\n",
      100*(double)i/(double)num_frames);
  }
  log_gpu.close();

  /*TEST CPU*/
  ofstream log_cpu("/media/keshav/Keshav/Dropbox (MIT)/anim_cpu/log_cpu.txt");
  for (int i = 0; i < num_frames; i++){
    sprintf(filename, "/media/keshav/Keshav/Dropbox (MIT)/anim_cpu/i0001%02d.bmp", i);
    plot.set_frame_number(i, num_frames);
    t1 = clock();
    plot_frame_cpu(plot, filename);
    t2 = clock();
    log_cpu << 1000*(double)(t2-t1)/(CLOCKS_PER_SEC) << "\n";
    if (!(i%(num_frames/100)) && (argc-1)) printf("Done %2.2f%% on CPU.\n",
      100*(double)i/(double)num_frames);
  }
  log_cpu.close();

  delete[] filename;
  return 0;
}
